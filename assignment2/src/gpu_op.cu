#include "hip/hip_runtime.h"
#include "./c_runtime_api.h"
#include <cassert>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

int divupround(int a, int b) {
  if(a % b == 0) return a / b;
  return a / b + 1;
}

int64_t totallength(DLArrayHandle array) {
  int64_t length = 1;
  for(int i = 0; i < array->ndim; i++) {
    length *= array->shape[i];
  }
  return length;
}

/* TODO: Your code here */
/* all your GPU kernel code, e.g. matrix_softmax_cross_entropy_kernel */

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)
__global__ void matrix_softmax_cross_entropy_kernel(int nrow, int ncol,
                                                    const float *input_a,
                                                    const float *input_b,
                                                    float *output) {
  // Dynamic shared memory, size provided at kernel launch.
  extern __shared__ float loss_per_row[];
  // Two dimensional thread blocks.
  int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
          threadIdx.x;
  if (y >= nrow) {
    return;
  }
  input_a += y * ncol;
  input_b += y * ncol;
  float maxval = *input_a;
  // Find max for a row.
  for (int x = 1; x < ncol; ++x) {
    maxval = max(maxval, input_a[x]);
  }
  // Deduct by max for a row, and raise to exp.
  float sum = 0;
  for (int x = 0; x < ncol; ++x) {
    sum += exp(input_a[x] - maxval);
  }
  // Compute per-row loss.
  float loss = 0;
  for (int x = 0; x < ncol; ++x) {
    loss -= input_b[x] * log(exp(input_a[x] - maxval) / sum);
  }
  loss_per_row[y] = loss;
  __syncthreads();
  // Compute reduce_mean across rows.
  float mean_loss = 0;
  // Use a single thread to reduce mean across rows.
  if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
    for (int i = 0; i < nrow; ++i) {
      mean_loss += loss_per_row[i];
    }
    mean_loss /= nrow;
    output[0] = mean_loss;
  }
}

__global__ void matrix_softmax_kernel(int nrow, 
                                      int ncol, 
                                      const float *input_data, 
                                      float *output_data){
  // Two dimensional thread blocks.
  int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
          threadIdx.x;
  if (y >= nrow) {
    return;
  }
  input_data += y * ncol;
  output_data += y * ncol;
  float maxval = *input_data;
  // Find max for a row.
  for (int x = 1; x < ncol; ++x) {
    maxval = max(maxval, input_data[x]);
  }
  // Deduct by max for a row, and raise to exp.
  float sum = 0;
  for (int x = 0; x < ncol; ++x) {
    sum += exp(input_data[x] - maxval);
  }

  for (int x = 0; x < ncol; ++x) {
    output_data[x] = exp(input_data[x] - maxval) / sum;
  }
}

__global__ void relu_kernel(int64_t length, 
                            const float *input_data,
                            float *output_data){
    int y = blockIdx.x * blockDim.x + threadIdx.x;
    if (y > length){
      return;
    }
    output_data[y] = max(0.0f, input_data[y]);
}

__global__ void relu_gradient_kernel(int64_t length,
                                     const float *input_data,
                                     const float *in_grad_data,
                                     float *output_data){
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  if (y > length){
    return;
  }
  output_data[y] = input_data[y]>0.0f? in_grad_data[y]:0.0f;
}

__global__ void array_set_kernel(int64_t length,
                                 float *array_data,
                                 float value){
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  if (y > length){
    return;
  }
  array_data[y] = value;
}

__global__ void broadcast_kernel(int64_t length, 
                                 const float *input_data,
                                 float *output_data){
  output_data += length * blockIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if(x < length) {
    output_data[x] = input_data[x];
  }
}

__global__ void reduce_sum_axis_zero_kernel(int64_t output_length, int reduce_size, const float* input, float *output) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  if(x >= output_length) return;
  float value = 0;
  for(int i = threadIdx.y; i < reduce_size; i+= blockDim.y) {
    value += input[i * output_length + x];
  }
  atomicAdd(output + x, value);
}

__global__ void matrix_add_kernel(int64_t length,
                                  const float *matA_data, 
                                  const float *matB_data, 
                                  float *output_data){
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  if (y > length){
    return;
  }
  output_data[y] = matA_data[y] + matB_data[y]; 
}

__global__ void matrix_add_by_const_kernel(int64_t length,
                                           const float *input_data,
                                           float val,
                                           float *output_data){
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  if (y > length){
    return;
  }
  output_data[y] = input_data[y] + val; 
}

__global__ void matrix_mul_kernel(int64_t length,
                                  const float *matA_data, 
                                  const float *matB_data, 
                                  float *output_data){
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  if (y > length){
    return;
  }
  output_data[y] = matA_data[y] * matB_data[y]; 
}

__global__ void matrix_mul_by_const_kernel(int64_t length,
                                           const float *input_data,
                                           float val,
                                           float *output_data){
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  if (y > length){
    return;
  }
  output_data[y] = input_data[y] * val; 
}

hipblasHandle_t cublas_handle = NULL;

int DLGpuArraySet(DLArrayHandle arr, float value) { /* TODO: Your code here */
  int64_t length = totallength(arr);
  float *array_data = (float *)arr->data;
  dim3 DimGrid((length-1)/1024+1, 1, 1);
  dim3 DimBlock(1024, 1, 1);
  array_set_kernel<<<DimGrid, DimBlock>>>(length, array_data, value);  
  return 0;
}

int DLGpuBroadcastTo(const DLArrayHandle input, DLArrayHandle output) {
  /* TODO: Your code here */
  int64_t length = totallength(input);
  const float *input_data = (float *)input->data;
  float *output_data = (float *)output->data;
  dim3 threads;
  if (length < 1024){
    threads.x = (int)length;
  }else{
    threads.x = 1024;
    threads.y = (int)((length + 1023)/1024);
  }
  broadcast_kernel<<<dim3(divupround(length, 1024), output->shape[0]), 1024>>>(length, input_data, output_data);
  return 0;
}

int DLGpuReduceSumAxisZero(const DLArrayHandle input, DLArrayHandle output) {
  /* TODO: Your code here */
  DLGpuArraySet(output, 0);
  int output_length = totallength(output);
  reduce_sum_axis_zero_kernel<<<divupround(output_length, 64), dim3(min(64, output_length), 16)>>>(output_length, input->shape[0], (float*)input->data, (float*)output->data);
  return 0;
}

int DLGpuMatrixElementwiseAdd(const DLArrayHandle matA,
                              const DLArrayHandle matB, DLArrayHandle output) {
  /* TODO: Your code here */
  int64_t length = totallength(output);
  const float *matA_data = (const float *)matA->data;
  const float *matB_data = (const float *)matB->data;
  float *output_data = (float *)output->data;
  dim3 DimGrid((length-1)/1024+1, 1, 1);
  dim3 DimBlock(1024, 1, 1);
  matrix_add_kernel<<<DimGrid, DimBlock>>>(length, matA_data, matB_data, output_data);
  return 0;
}

int DLGpuMatrixElementwiseAddByConst(const DLArrayHandle input, float val,
                                     DLArrayHandle output) {
  /* TODO: Your code here */
  int64_t length = totallength(output);
  const float *input_data = (const float *)input->data;
  float *output_data = (float *)output->data;
  dim3 DimGrid((length-1)/1024+1, 1, 1);
  dim3 DimBlock(1024, 1, 1);
  matrix_add_by_const_kernel<<<DimGrid, DimBlock>>>(length, input_data, val, output_data);
  return 0;
}

int DLGpuMatrixElementwiseMultiply(const DLArrayHandle matA,
                                   const DLArrayHandle matB,
                                   DLArrayHandle output) {
  /* TODO: Your code here */
  int64_t length = totallength(output);
  const float *matA_data = (const float *)matA->data;
  const float *matB_data = (const float *)matB->data;
  float *output_data = (float *)output->data;
  dim3 DimGrid((length-1)/1024+1, 1, 1);
  dim3 DimBlock(1024, 1, 1);
  matrix_mul_kernel<<<DimGrid, DimBlock>>>(length, matA_data, matB_data, output_data);
  return 0;
}

int DLGpuMatrixMultiplyByConst(const DLArrayHandle input, float val,
                               DLArrayHandle output) {
  /* TODO: Your code here */
  int64_t length = totallength(output);
  const float *input_data = (const float *)input->data;
  float *output_data = (float *)output->data;
  dim3 DimGrid((length-1)/1024+1, 1, 1);
  dim3 DimBlock(1024, 1, 1);
  matrix_mul_by_const_kernel<<<DimGrid, DimBlock>>>(length, input_data, val, output_data);
  return 0;
}

int DLGpuMatrixMultiply(const DLArrayHandle matA, bool transposeA,
                        const DLArrayHandle matB, bool transposeB,
                        DLArrayHandle matC) {
  /* TODO: Your code here */
  // Hint: use cublas
  // cublas assume matrix is column major
  if(!cublas_handle) {
    hipblasCreate(&cublas_handle);
  }

  float one = 1.0f;
  float zero = 0.0f;
  int m = matC->shape[1];
  int n = matC->shape[0];
  int k = transposeA ? matA->shape[0] : matA->shape[1];

  hipblasSgemm(cublas_handle,
    transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
    transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
    m, n, k,
    &one,
    (const float*)matB->data, !transposeB ? m : k,
    (const float*)matA->data, !transposeA ? k : n,
    &zero,
    (float*)matC->data, m
  );
  return 0;
}

int DLGpuRelu(const DLArrayHandle input, DLArrayHandle output) {
  /* TODO: Your code here */
  int64_t length =  totallength(output);
  const float *input_data = (const float *)input->data;
  float *output_data = (float *)output->data;
  dim3 DimGrid((length-1)/1024+1, 1, 1);
  dim3 DimBlock(1024, 1, 1);
  relu_kernel<<<DimGrid, DimBlock>>>(length, input_data, output_data);
  return 0;
}

int DLGpuReluGradient(const DLArrayHandle input, const DLArrayHandle in_grad,
                      DLArrayHandle output) {
  /* TODO: Your code here */
  int64_t length =  totallength(output);
  const float *input_data = (const float *)input->data;
  const float *in_grad_data = (const float *)in_grad->data;
  float *output_data = (float *)output->data;
  dim3 DimGrid((length-1)/1024+1, 1, 1);
  dim3 DimBlock(1024, 1, 1); 
  relu_gradient_kernel<<<DimGrid, DimBlock>>>(length, input_data, in_grad_data, output_data);
  return 0;
}

int DLGpuSoftmax(const DLArrayHandle input, DLArrayHandle output) {
  /* TODO: Your code here */
  //assert(input->ndim == 2);
  //assert(output->ndim == 1);
  int nrow = input->shape[0];
  assert(nrow <= 1024 * 4);
  int ncol = input->shape[1];
  const float *input_data = (const float *)input->data;
  float *output_data = (float *)output->data;
  dim3 threads;
  if (nrow < 1024){
    threads.x = nrow;
  }else{
    threads.x = 1024;
    threads.y = (nrow + 1023)/1024;
  }
  matrix_softmax_kernel<<<1, threads, nrow * sizeof(float)>>>(nrow, ncol, input_data, output_data);
  return 0;
}

int DLGpuSoftmaxCrossEntropy(const DLArrayHandle input_a,
                             const DLArrayHandle input_b,
                             DLArrayHandle output) {
  assert(input_a->ndim == 2);
  assert(input_b->ndim == 2);
  assert(output->ndim == 1);
  assert(input_a->shape[0] == input_b->shape[0] &&
         input_a->shape[1] == input_b->shape[1]);
  int nrow = input_a->shape[0];
  // Maximum x- or y-dimension of a block = 1024
  // But we need 'nrow' shared memory, and max shared memory is 48KB.
  // Conservatively allow max 16KB shared memory.
  assert(nrow <= 1024 * 4);
  int ncol = input_a->shape[1];
  const float *input_data_a = (const float *)input_a->data;
  const float *input_data_b = (const float *)input_b->data;
  float *output_data = (float *)output->data;
  dim3 threads;
  if (nrow <= 1024) {
    threads.x = nrow;
  } else {
    threads.x = 1024;
    threads.y = (nrow + 1023) / 1024;
  }
  // 1 block, each block with 'threads' number of threads with 'nrow' shared
  // memory size
  matrix_softmax_cross_entropy_kernel<<<1, threads, nrow * sizeof(float)>>>(
      nrow, ncol, input_data_a, input_data_b, output_data);
  return 0;
}
